#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_vector_types.h>
#include <math.h>
#include <cutil.h>
#include "precision.h"

// includes, kernels
#include <HCq_kernel.cu>

extern "C" {
	
	int gethostname(char *name, size_t len);
	void bzero(void *s, size_t n);
	char *strcpy(char *dest, const char *src);
	
#define MAXDEVICES 4
#define MAXNODES 16
	int cstGPU_init (float* obs_h, int mObs)
	{
		int myproc=0;
		int mydevice=0;
		
		int i, dc;
		hipError_t cerr ;
		char hostname[64] ;
		struct hipDeviceProp_t dp ;
		//  manage devices if multiheaded
		hipGetDeviceCount( &dc ) ;
		if ( dc > MAXDEVICES ) 
		{ fprintf(stderr, "warning: more than %d devices on node (%d)\n", MAXDEVICES, dc ) ; dc = MAXDEVICES ; }
		fprintf(stderr,"Number of devices on this node: %d\n", dc) ;
		
		// i = *myproc % dc ;
		
		i = mydevice ;
		if ( dc > 0 ) 
		{
			cerr = hipSetDevice( i );
			if ( cerr ) {
				fprintf(stderr,"    non-zero cerr %d\n",cerr) ;
			}
		}
		gethostname( hostname, 64 ) ;
		fprintf(stderr,"Setting device %02d for task %03d on host %s\n",i,myproc,hostname) ;
		
		cerr = hipGetDeviceProperties( &dp, i );
		if ( cerr ) {
			fprintf(stderr,"Device %02d: cerr = %d\n",i,cerr) ;
		} else {
			fprintf(stderr,"Device %02d: name %s\n",i,dp.name) ;
			fprintf(stderr,"Device %02d: mem       %d\n",i,(int)dp.totalGlobalMem) ;
			fprintf(stderr,"Device %02d: smem      %d\n",i,(int)dp.sharedMemPerBlock) ;
			fprintf(stderr,"Device %02d: nreg      %d\n",i,dp.regsPerBlock) ;
			fprintf(stderr,"Device %02d: warp      %d\n",i,dp.warpSize) ;
			fprintf(stderr,"Device %02d: pitch     %d\n",i,(int)dp.memPitch) ;
			fprintf(stderr,"Device %02d: maxthrds  %d\n",i,dp.maxThreadsPerBlock) ;
			fprintf(stderr,"Device %02d: maxtdim   %d %d %d\n",i,dp.maxThreadsDim[0]
					,dp.maxThreadsDim[1]
					,dp.maxThreadsDim[2]) ;
			fprintf(stderr,"Device %02d: maxgdim   %d %d %d\n",i,dp.maxGridSize[0]
					,dp.maxGridSize[1]
					,dp.maxGridSize[2]) ;
			fprintf(stderr,"Device %02d: clock     %d\n",i,dp.clockRate) ;
			fprintf(stderr,"Device %02d: talign    %d\n",i,(int)dp.textureAlignment) ;
		}
				
		float BoundaryConditions[9][4] = 
		//	0		1		M-1		M
		{{	-4,		-1,		-1,		-4 },
		{	0,		1,		1,		0 },
		{	2,		-1,		-1,		2 },
		{   -4,     -1,     1,      0 },
		{   -4,     -1,     -1,     2 },
		{   0,      1,      -1,     -4 },
		{   0,      1,      -1,     2 },
		{   2,      -1,     -1,     -4 },
		{   2,      -1,     1,      0 }};

		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(BC), BoundaryConditions, sizeof(BoundaryConditions)));

		CUDA_SAFE_CALL(hipMalloc((void **)&obs_d,mObs*9*sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpy(obs_d,obs_h,mObs*9*sizeof(float),hipMemcpyHostToDevice)) ;

		CUDA_SAFE_CALL(hipMalloc((void **)&HCq_d,mObs*sizeof(float)));
				
		return(0) ;
	}
	
	// Preload the coefficients 
	void loadSplineCoeffs_GPU(float* coeffHost, int numCoeffs)
	{
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(coeffDevice),coeffHost,numCoeffs*sizeof(float))) ;
	}
	
	void cstGPU_finalize()
	{
		// clean up memory
		CUDA_SAFE_CALL(hipFree(obs_d));
		CUDA_SAFE_CALL(hipFree(HCq_d));
	}
	
	// Try and evaluate a spline on the GPU
	void HCq_GPU(int mObs, float rmax, float rmin, float zmax, float zmin, float* HCq_h, int pState, int zState)
	{
		
		// Constants and boundary conditions
		int R = pState-1;
		float dr = (rmax - rmin) / R;
		float drrecip = 1./dr;
		int Z = zState-1;
		float dz = (zmax - zmin) / Z;
		float dzrecip = 1./dz;
		float onesixth = 1./6.;
		
		/* create and start timer
		unsigned int timer = 0;
		CUT_SAFE_CALL(cutCreateTimer(&timer));
		CUT_SAFE_CALL(cutStartTimer(timer)); */
		
		// setup execution parameters
		int rem = mObs%BLOCKSIZE != 0 ? 1 : 0;
		dim3 threads(BLOCKSIZE);
		dim3 grid(mObs / BLOCKSIZE + rem);
		
		// execute the kernel
		HCq_kernel<<< grid, threads >>>(obs_d, HCq_d, R, Z, rmin, dr, drrecip, zmin, dz, dzrecip, onesixth);
		
		// check if kernel execution generated and error
		CUT_CHECK_ERROR("Kernel execution failed");
		
		// copy result from device to host
		CUDA_SAFE_CALL(hipMemcpy(HCq_h, HCq_d, mObs*sizeof(float), hipMemcpyDeviceToHost));
		
		/* stop and destroy timer
		CUT_SAFE_CALL(cutStopTimer(timer));
		printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
		CUT_SAFE_CALL(cutDeleteTimer(timer)); */
		
	}
	
}
